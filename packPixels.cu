
#include <hip/hip_runtime.h>
typedef struct { float r, g, b, a; } Color;

__global__
void packPixels(Color* RGBInput, Color* AlphaInput, Color* Output){
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;

    Output[index].r = RGBInput[index].r;
    Output[index].g = RGBInput[index].g;
    Output[index].b = RGBInput[index].b;

    Output[index].a = AlphaInput[index].r;
}

__global__
void packPixels4(){

}

extern "C"
{
    __declspec(dllexport) Color* PackPixels(Color* RGBTexture, Color* AlphaTexture, int sizeX, int sizeY);

    //float4 RGBInput, AlphaInput, Output;
    bool isSquare;
    bool XBiggerThanY;
    int tileCount, tileSize;
    size_t bufferSize;
    Color* OutBuffer;

    Color* PackPixels(Color* RGBTexture, Color* AlphaTexture, int sizeX, int sizeY){

        // Some checks that will be useful to calculate launch parameters
        if (sizeX == sizeY){
            isSquare = true;
            XBiggerThanY = false;
            tileCount = 1;
            tileSize = sizeX;
        } else if (sizeX > sizeY){
            isSquare = false;
            XBiggerThanY = true;
            tileCount = sizeX / sizeY;
            tileSize = sizeY;
        } else {
            isSquare = false;
            XBiggerThanY = false;
            tileCount = sizeY / sizeX;
            tileSize = sizeX;
        }

        bufferSize = sizeX * sizeY * sizeof(Color);
        Color *RGBInput, *AlphaInput, *Output;

        // Allocate the textures on GPU memory
        // and the output on host
        hipMalloc(&RGBInput, bufferSize);
        hipMalloc(&AlphaInput, bufferSize);
        hipMalloc(&Output, bufferSize);
        hipHostMalloc(&OutBuffer, bufferSize, hipHostMallocDefault);

        // Copy color values to GPU memory
        hipMemcpy(RGBInput, RGBTexture, bufferSize, hipMemcpyHostToDevice);
        hipMemcpy(AlphaInput, AlphaTexture, bufferSize, hipMemcpyHostToDevice);

        dim3 grid, block;

        block = {32, 32};
        unsigned int blocksInTileSize = tileSize / 32;
        grid = {blocksInTileSize, blocksInTileSize};
        hipStream_t streams[tileCount];

        for (int i = 0; i < tileCount; i++){
            int offset = (i) * tileSize;
            size_t count = bufferSize/tileCount;
            hipMemcpyAsync(&RGBInput[offset], &RGBTexture[offset], count, hipMemcpyHostToDevice, streams[i]);
            hipMemcpyAsync(&AlphaInput[offset], &AlphaTexture[offset], count, hipMemcpyHostToDevice, streams[i]);

            packPixels<<<grid, block, 0, streams[i]>>>(RGBInput, AlphaInput, Output);
            hipMemcpyAsync(&OutBuffer[offset], &Output[offset], count, hipMemcpyDeviceToHost, streams[i]);
        }

        hipDeviceSynchronize();

        hipFree(RGBInput);
        hipFree(AlphaInput);
        hipFree(Output);

        return OutBuffer;
    }

    void ClearData(){

    }
}